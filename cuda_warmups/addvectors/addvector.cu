// cuda_example.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add_kernel(int *a, int *b, int *c, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

int main() {
  int n = 10;
  int *a, *b, *c;

  hipMalloc(&a, n * sizeof(int));
  hipMalloc(&b, n * sizeof(int));
  hipMalloc(&c, n * sizeof(int));

  for (int i = 0; i < n; i++) {
    a[i] = i;
    b[i] = i * 2;
  }

  add_kernel<<<1, 1>>>(a, b, c, n);

  hipFree(a);
  hipFree(b);
  hipFree(c);

  for (int i = 0; i < n; i++) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  return 0;
}