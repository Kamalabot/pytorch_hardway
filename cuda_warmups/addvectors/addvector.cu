// cuda_example.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void add_kernel(int *a, int *b, int *c, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  printf("%d is entered \n", i);
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

int main() {
  int n = 10;
  int *a, *b, *c;

  hipMallocManaged(&a, n * sizeof(int));
  hipMallocManaged(&b, n * sizeof(int));
  hipMallocManaged(&c, n * sizeof(int));

  for (int i = 0; i < n; i++) {
    a[i] = i;
    b[i] = i * 2;
    cout << "a[i]: "<< a[i] * 2 << endl;
  }

  // add_kernel<<<1, 1>>>(a, b, c, n);
  add_kernel<<<1, 10>>>(a, b, c, n);
  hipDeviceSynchronize();

  for (int i = 0; i < n; i++) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }
  
  hipFree(a);
  hipFree(b);
  hipFree(c);

  return 0;
}