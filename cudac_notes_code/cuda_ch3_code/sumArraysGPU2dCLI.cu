#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>

using namespace std;
// used for checking the errors in the function calls and print them

#define CHECK(call)                                                                 \
{                                                                                   \
    const hipError_t error = call;                                                 \
    if (error != hipSuccess)                                                       \
    {                                                                               \
        cout << "code: " << error << "reason: " << hipGetErrorString(error) << endl;\
        exit(1);                                                                    \
    }                                                                               \
}

// Comparing the result of matrix operation by host function and kernel
void checkResult(float *hostRef, float *gpuRef, const int N){
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i = 0; i < N; i++){
        if(abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = 0;
            cout << "Arrays do not match." << endl;
            cout << "host: " << hostRef[i] << "gpu: " << gpuRef[i] << endl;
            break;
        }
    }
    if (match) cout << "Arrays Match" << endl;
}

void initialData(float *ip, int size){
    time_t t;  // t is of time time_t, and its address is sent to srand
    // srand((unsigned int) time(&t)); // the returned time_t value is casted
    srand(static_cast<unsigned int>(time(0)));
    for (int j=0; j < size; j++){
        ip[j] = (float) ( rand() & 0xFF ) / 10.0f;
    }
}

int cpuSecond(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)tp.tv_sec + (double)tp.tv_usec * 1.e-6;
}

__global__ void sumArrayOnGPU2d(float *A, float *B, float *C, int nx, int ny){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; // calc ix from the ids of threads & blocks
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y; // calc iy from the ids of blocks & threads
    unsigned int idx = iy * nx + ix;  // calc the id of the array from ix & iy
    if(ix < nx && iy < ny){
        C[idx] = A[idx] + B[idx];  // the 2d matrix is linearly placed in the memory
    }
}

int main(int argc, char* argv[]){
    int dev = 0; // setup device to be 0
    hipSetDevice(dev);

    int nx = 1 << 14; // set data 16,384 elems
    int ny = 1 << 14; // set data 16,384 elems
    cout << "Vector x size: " << nx << endl;
    cout << "Vector y size: " << ny << endl;
    
    int nxy = nx * ny;
    size_t nBytes = nxy * sizeof(float); 

    float *h_A, *h_B, *hostRef, *gpuRef;
    // https://www.geeksforgeeks.org/malloc-vs-new/ 
    // we can implement new based memory allocation 
    h_A = (float *)malloc(nBytes);  // its going to linear memory alloc
    h_B = (float *)malloc(nBytes);

    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    initialData(h_A, nxy);
    initialData(h_B, nxy);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A , nBytes);
    // &d_A is address in device memory, which is holding the 
    // pointer to the array of data
    hipMalloc((float **)&d_B , nBytes);
    hipMalloc((float **)&d_C , nBytes);

    // move data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // int dimx = 32; // 32, 16
    // int dimy = 32; // 16, 16
    // invoke kernel at host side
    if (argc > 2) {
        dimx = atoi(arvg[1]);
        dimy = atoi(arvg[2]);
    }
    dim3 block (dimx, dimy);
    dim3 grid ((nx + block.x - 1) / block.x, (ny + block.y -1) / block.y);

    double iSt = cpuSecond();    
    sumArrayOnGPU2d<<<grid, block>>>(d_A, d_B, d_C, nx, ny);
    double iEl = cpuSecond() - iSt;

    cout << "grid.x " << grid.x << "block.x " << block.x << endl;
    printf("sumArrayOnGPU2d<<<(%d, %d), (%d, %d)>>> elapsed %f sec. \n",
          grid.x, grid.y, block.x, block.y, iEl);

    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    sumArrayOnHost(h_A, h_B, hostRef, nxy);

    checkResult(hostRef, gpuRef, nxy);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}
