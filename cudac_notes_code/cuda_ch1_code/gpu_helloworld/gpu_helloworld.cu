
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void helloGPU(void){
    printf("Hello from GPU...");
}

int main(){
    cout << "Hello from CPU" << endl;
    helloGPU <<<1, 10>>>();
    hipDeviceReset(); // with this one version
    // cudaDeviceSynchronize(); // with this 2nd version
    return 0;
}