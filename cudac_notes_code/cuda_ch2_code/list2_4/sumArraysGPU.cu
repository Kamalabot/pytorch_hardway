#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
// used for checking the errors in the function calls and print them
#define CHECK(call)                                                                 \
{                                                                                   \
    const hipError_t error = call;                                                 \
    if (error != hipSuccess)                                                       \
    {                                                                               \
        cout << "code: " << error << "reason: " << hipGetErrorString(error) << endl;\
        exit(1);                                                                    \
    }                                                                               \
}

// Comparing the result of matrix operation by host function and kernel
void checkResult(float *hostRef, float *gpuRef, const int N){
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i = 0; i < N; i++){
        if(abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = 0;
            cout << "Arrays do not match." << endl;
            cout << "host: " << hostRef[i] << "gpu: " << gpuRef[i] << endl;
            break;
        }
    }
    if (match) cout << "Arrays Match" << endl;
}
// summing array on host
void sumArrayOnHost(float *a, float *b, float *c, const int N){
    // enumerate and sum
    for(int idx = 0; idx < N; idx++){
        c[idx] = a[idx] + b[idx]; // just take two elements and add them
    }
}

void initialData(float *ip, int size){
    // time_t t;  // t is of time time_t, and its address is sent to srand
    // srand((unsigned int) time(&t)); // the returned time_t value is casted
    srand(static_cast<unsigned int>(time(0)));
    for (int j=0; j < size; j++){
        ip[j] = (float) ( rand() & 0xFF ) / 10.0f;
    }
}

__global__ void sumArrayOnGPU(float *A, float *B, float *C){
    int i = threadIdx.x; // getting the ids of initialized threads
    // int i = blockIdx.x; // if the threadIdx.x is set to 1
    C[i] = A[i] + B[i];
}

__global__ void gensumArrayOnGPU(float *A, float *B, float *C){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

int main(){
    int dev = 0; // setup device to be 0
    hipSetDevice(dev);

    int nElem = 32; // set data
    cout << "Vector size: " << nElem << endl;

    size_t nBytes = nElem * sizeof(float); 

    float *h_A, *h_B, *hostRef, *gpuRef;
    // https://www.geeksforgeeks.org/malloc-vs-new/ 
    // we can implement new based memory allocation 
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);

    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    // Memset() is a C++ library function used to
    // fill a memory block with a particular value. This function
    // takes three arguments: a pointer to the starting address of the
    // memory block to be filled, the value to be set, and the number 
    // of bytes to be filled.
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A , nBytes);
    // &d_A is address in device memory, which is holding the 
    // pointer to the array of data
    hipMalloc((float **)&d_B , nBytes);
    hipMalloc((float **)&d_C , nBytes);

    // move data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // invoke kernel at host side
    dim3 block (nElem);
    dim3 grid (nElem/block.x);
    // in case of 32 / 32, there will be 1 block
    // if the threadIdx.x is less than 32, then blocks increases to 2
    sumArrayOnGPU<<<grid, block>>>(d_A, d_B, d_C);
    cout << "grid.x " << grid.x << "block.x " << block.x << endl;

    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    sumArrayOnHost(h_A, h_B, hostRef, nElem);

    checkResult(hostRef, gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}