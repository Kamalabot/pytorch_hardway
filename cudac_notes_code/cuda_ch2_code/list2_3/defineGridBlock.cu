#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main(){
    int nElem = 1024;
    dim3 block (1024);
    dim3 grid ((nElem + block.x - 1)/block.x);
    cout << "block.x: " << block.x << "grid.x: " << grid.x << endl;

    block.x = 512;
    grid.x = (nElem + block.x - 1)/block.x;
    cout << "block.x: " << block.x << "grid.x: " << grid.x << endl;

    block.x = 256;
    grid.x = (nElem + block.x - 1)/block.x;
    cout << "block.x: " << block.x << "grid.x: " << grid.x << endl;

    block.x = 128;
    grid.x = (nElem + block.x - 1)/block.x;
    cout << "block.x: " << block.x << "grid.x: " << grid.x << endl;

}