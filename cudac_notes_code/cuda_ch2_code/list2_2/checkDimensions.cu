#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void checkDim(void){
    printf("threadIdx: (%d, %d, %d) blockIdx: (%d, %d, %d) blockDim: (%d, %d, %d) gridDim: (%d, %d, %d) ",
        threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, 
        blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
        // all four internal variables that are initialized by the kernel are printed
}

int main(){
    int nElem = 6;
    dim3 block (3);
    dim3 grid ((nElem + block.x - 1)/block.x);

    // printf("grid.x %d grid.y %d grid.z %d", grid.x, grid.y, grid.z);
    cout << "grid.x  grid.y grid.z" << (grid.x, grid.y, grid.z) << endl;
    // printf("block.x %d block.y %d block.z %d", block.x, block.y, block.z);
    cout << "block.x, block.y, block.z" << (block.x, block.y, block.z) << endl;

    checkDim<<<grid, block>>>();

    hipDeviceReset();

    return 0;
}